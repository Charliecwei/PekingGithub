#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define BLOCK_WIDTH 16
#define TILE_WIDTH  BLOCK_WIDTH

extern "C" void gpu_mat_mul(float* h_M, float* h_N, float* h_P, int Mwidth,int Nwidth,int Swidth);

__global__
void gpu_mat_mul_kernel(float* M, float* N, float* P, int Mwidth, int Nwidth, int Swidth){

  __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x; 
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
 // printf("blockx is %d,blocky is%d\n",bx,by);
  // Identify the row and column of the P element to work on
  // Each thread works on an element of P
  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;
//  printf("Row %d,Col is%d\n",Row,Col);
  float sum = 0;
  int phase_num = (int)ceil((double) Nwidth/TILE_WIDTH);
  // Each thread loads 'Row'th row of M and 'Col'th column of N
  for (int ph = 0; ph < phase_num-1; ph++) {    

    // Collaboratively load data into shared memory
    Mds[ty][tx] = M[Row * Nwidth + ph * TILE_WIDTH + tx];   
    Nds[ty][tx] = N[(ph * TILE_WIDTH + ty) * Swidth + Col];

    __syncthreads();
    for (int k = 0; k < TILE_WIDTH; k++) { 
      sum += Mds[ty][k] * Nds[k][tx];
    }
    __syncthreads();
  }
  int ph = phase_num-1;
  int res = Nwidth - ph*TILE_WIDTH;
 if (tx < res){
      Mds[ty][tx] = M[Row * Nwidth + ph * TILE_WIDTH + tx];
  }
if (ty < res){
    Nds[ty][tx] = N[(ph * TILE_WIDTH + ty) * Swidth + Col];
  }
   __syncthreads();//Barrier
      for (int k = 0; k < res ; k++) {
        sum += Mds[ty][k] * Nds[k][tx];
      }
      __syncthreads();

  if (Row<Mwidth&&Col<Swidth){
  P[Row * Swidth + Col] = sum;
  }
}
void gpu_mat_mul(float* h_M, float* h_N, float* h_P, int Mwidth,int Nwidth,int Swidth) {
  float *d_M, *d_N, *d_P;

  size_t size_of_float = sizeof(float);
  size_t size_M = Mwidth * Nwidth * size_of_float;
  size_t size_N = Nwidth * Swidth * size_of_float;
  size_t size_P = Mwidth * Swidth * size_of_float;

  hipMalloc((void**)&d_M, size_M);
  hipMalloc((void**)&d_N, size_N);
  hipMalloc((void**)&d_P, size_P);
    
  hipMemcpy(d_M, h_M, size_M, hipMemcpyHostToDevice);
  hipMemcpy(d_N, h_N, size_N, hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  float elapsed_time = 0.0;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  dim3 grid_dim((int)ceil((double)Swidth/BLOCK_WIDTH),(int)ceil((double) Mwidth/BLOCK_WIDTH), 1);
  dim3 block_dim(BLOCK_WIDTH, BLOCK_WIDTH, 1);
  gpu_mat_mul_kernel<<<grid_dim, block_dim>>>(d_M, d_N, d_P, Mwidth,Nwidth,Swidth);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipMemcpy(h_P, d_P, size_P, hipMemcpyDeviceToHost);
    
  // Free device memory for M, N, P
  hipFree(d_M);
  hipFree(d_N);
  hipFree(d_P);

  hipEventElapsedTime(&elapsed_time, start, stop);
    
  printf("  grid  dim:  %d, %d, %d.\n", grid_dim.x, grid_dim.y, grid_dim.z);
  printf("  block dim: %d, %d, %d.\n", block_dim.x, block_dim.y, block_dim.z);
  printf("  kernel time: %.5f sec\n", elapsed_time / 1000);

  hipEventDestroy(start);
  hipEventDestroy(stop);
}


